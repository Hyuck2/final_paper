#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

torch::Tensor forward(torch::Tensor input, std::vector<torch::Tensor> parameter){
    torch::Tensor output;
    
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward, "cnn forward (CUDA)");
    //m.def("backward", &backward, "cnn backward (CUDA)");
  }